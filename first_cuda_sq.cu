#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define DATA_SIZE 1048576

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}

__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{
    int sum = 0;
    int i;
    clock_t start = clock();
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }

    *result = sum;
    *time = clock() - start;
}

int main()
{
    if(!InitCUDA()) {
        return 0;
    }

    printf("CUDA initialized.\n");

    int data[DATA_SIZE];

    GenerateNumbers(data, DATA_SIZE);
    int* gpudata, *result;
    clock_t* time;
    clock_t start_g, stop_g;
    start_g = clock();
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int));
    hipMalloc((void**) &time, sizeof(clock_t));
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares<<<1, 1, 0>>>(gpudata, result, time);

    int sum;
    clock_t time_used;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
    hipFree(time);
    stop_g = (clock() - start_g);

    printf("sum (GPU): %d time: %f timeg: %f \n", sum, (double)time_used / CLOCKS_PER_SEC, (double) stop_g / CLOCKS_PER_SEC);

    clock_t start, stop;

    start = clock();
    sum = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i];
    }

    stop = clock() - start;
    printf("sum (CPU): %d time: %f \n", sum, (double)stop / CLOCKS_PER_SEC);

    return 0;
}
